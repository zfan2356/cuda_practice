#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <bit.h>

__device__ float devData;

__global__ void checkGlobalVar() {
    printf("Device: The value of the global variable is %f\n", devData);
    devData += 2.0;
}

int main() {
    float val = 3.14f;
    hipMemcpyToSymbol(HIP_SYMBOL(&devData), &val, sizeof(float));
    std::cout << "Host: copy" << std::endl;

    checkGlobalVar<<<1, 1>>>();

    hipMemcpyFromSymbol(&val, HIP_SYMBOL(&devData), sizeof(float));
    std::cout << val << std::endl;
    hipDeviceReset();
    return 0;
}