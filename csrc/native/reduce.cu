#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#define WarpSize 32
//latency: 1.254ms
template <int blockSize>
__device__ float WarpShuffle(float sum) {
    // 使用模板参数保持与调用处的一致性
    // 优化shfl指令顺序，保持注释清晰度
    sum += __shfl_down_sync(0xffffffff, sum, 16);
    sum += __shfl_down_sync(0xffffffff, sum, 8);
    sum += __shfl_down_sync(0xffffffff, sum, 4);
    sum += __shfl_down_sync(0xffffffff, sum, 2);
    sum += __shfl_down_sync(0xffffffff, sum, 1);
    return sum;
}

template <int blockSize>
__global__ void reduce_warp_level(float *d_in,float *d_out, unsigned int n){
    float sum = 0;//当前线程的私有寄存器，即每个线程都会拥有一个sum寄存器

    unsigned int tid = threadIdx.x;
    unsigned int gtid = blockIdx.x * blockSize + threadIdx.x;
    // 分配的线程总数
    unsigned int total_thread_num = blockSize * gridDim.x;
    // 基于v5的改进：不用显式指定一个线程处理2个元素，而是通过L30的for循环来自动确定每个线程处理的元素个数
    for (int i = gtid; i < n; i += total_thread_num)
    {
        sum += d_in[i];
    }
    
    // 用于存储partial sums for each warp of a block
    __shared__ float WarpSums[blockSize / WarpSize]; 
    // 当前线程在其所在warp内的ID
    const int laneId = tid % WarpSize;
    // 当前线程所在warp在所有warp范围内的ID
    const int warpId = tid / WarpSize; 
    // 对当前线程所在warp作warpshuffle操作，直接交换warp内线程间的寄存器数据
    sum = WarpShuffle<blockSize>(sum);
    if(laneId == 0) {
        WarpSums[warpId] = sum;
    }
    __syncthreads();
    //至此，得到了每个warp的reduce sum结果
    //接下来，再使用第一个warp(laneId=0-31)对每个warp的reduce sum结果求和
    //首先，把warpsums存入前blockDim.x / WarpSize个线程的sum寄存器中
    //接着，继续warpshuffle
    sum = (tid < blockSize / WarpSize) ? WarpSums[tid] : 0;
    // Final reduce using first warp
    if (warpId == 0) {
        sum = WarpShuffle<blockSize/WarpSize>(sum); 
    }
    // store: 哪里来回哪里去，把reduce结果写回显存
    if (tid == 0) {
        d_out[blockIdx.x] = sum;
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //int GridSize = 100000;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_warp_level<blockSize><<<Grid,Block>>>(d_a, d_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d \n", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < GridSize;i++){
            printf("resPerBlock : %lf ",out[i]);
        }
        printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_warp_level latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}