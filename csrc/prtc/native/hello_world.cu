#include "hip/hip_runtime.h"
#include <bit.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

__device__ float devData;

__global__ void checkGlobalVar() {
  printf("Device: The value of the global variable is %f\n", devData);
  devData += 2.0;
}

int main() {
  float val = 3.14f;
  hipMemcpyToSymbol(HIP_SYMBOL(&devData), &val, sizeof(float));
  std::cout << "Host: copy" << std::endl;

  checkGlobalVar<<<1, 1>>>();

  hipMemcpyFromSymbol(&val, HIP_SYMBOL(&devData), sizeof(float));
  std::cout << val << std::endl;
  hipDeviceReset();
  return 0;
}
