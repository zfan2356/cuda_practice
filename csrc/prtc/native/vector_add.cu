#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda/atomic>


constexpr int ARRAY_SIZE = 1E8;
constexpr int MEMORY_OFFSET = 1E7;
constexpr int BENCH_ITER = 10;
constexpr int THREADS_NUM = 256;

__global__ void mem_bw (float* A,  float* B, float* C){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	for(int i = idx; i < MEMORY_OFFSET / 4; i += blockDim.x * gridDim.x) {
		// printf("index = %d, i = %d\n", idx, i);
		float4 a1 = reinterpret_cast<float4*>(A)[i];
		float4 b1 = reinterpret_cast<float4*>(B)[i];
		float4 c1;
		c1.x = a1.x + b1.x;
		c1.y = a1.y + b1.y;
		c1.z = a1.z + b1.z;
		c1.w = a1.w + b1.w;
		reinterpret_cast<float4*>(C)[i] = c1;
	}
}

void vec_add_cpu(float *x, float *y, float *z, int N) {
    for (int i = 0; i < 20; i++) z[i] = y[i] + x[i];
}



int main(){
	float *A = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *B = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *C = (float*) malloc(ARRAY_SIZE*sizeof(float));

	float *A_g;
	float *B_g;
	float *C_g;

	float milliseconds = 0;

	for (uint32_t i = 0; i < ARRAY_SIZE; i++) {
		A[i] = (float)i;
		B[i] = (float)i;
	}

	hipMalloc((void**)&A_g, ARRAY_SIZE * sizeof(float));
	hipMalloc((void**)&B_g, ARRAY_SIZE * sizeof(float));
	hipMalloc((void**)&C_g, ARRAY_SIZE * sizeof(float));

	hipMemcpy(A_g, A, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_g, B, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  
	int BlockNums = MEMORY_OFFSET / 256;
    //warm up to occupy L2 cache
	printf("warm up start\n");
	mem_bw<<<BlockNums / 4, THREADS_NUM>>>(A_g, B_g, C_g);
	printf("warm up end\n");
    // time start using cudaEvent
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	for (int i = BENCH_ITER - 1; i >= 0; --i) {
		mem_bw<<<BlockNums / 4, THREADS_NUM>>>(A_g + i * MEMORY_OFFSET, B_g + i * MEMORY_OFFSET, C_g + i * MEMORY_OFFSET);
	}
	// time stop using cudaEvent
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

	hipMemcpy(C, C_g, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	/* CPU compute */
	float* C_cpu_res = (float *) malloc(20 * sizeof(float));
	vec_add_cpu(A, B, C_cpu_res, ARRAY_SIZE);

	/* check GPU result with CPU*/
	for (int i = 0; i < 20; ++i) {
		/* 测量显存带宽时, 修改C_cpu_res[i]为0 */
		if (fabs(C_cpu_res[i] - C[i]) > 1e-6) {
			printf("Result verification failed at element index %d!\n", i);
		}
	}
	printf("Result right\n");

  	hipFree(A_g);
  	hipFree(B_g);
  	hipFree(C_g);

  	free(A);
  	free(B);
  	free(C);
  	free(C_cpu_res);
}