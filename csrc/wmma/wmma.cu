#include <>

__device__ void mma_simple(half *a, half *b, half *c) {
    using namespace nvcuda::wmma;
    fragment<matrix_a, 16, 16, 16, half, row_major> a_frag;
    fragment<matrix_b, 16, 16, 16, half, col_major> b_frag;
    fragment<accumulator, 16, 16, 16, half> c_frag;
}